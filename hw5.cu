/*
 Name: Anderson B. Liddle
 Email: abliddle@crimson.ua.edu
 Course Section: CS 581
 Homework # 5
 Instructions to compile the program: nvcc -O3 hw5.cu -o hw5
 Instructions to run the program: ./hw3 <dimensions (int)>
                                    <max_generations (int)>
                                    <block size (NxN)>
                                    <output directory (string)>
                                    <stagnationcheck (boolean 1 or 0)>
 Please use this format for testing: ./hw5 5000 5000 32 . 1
 GITHUB LINK - https://github.com/andersonbliddle/CS_581_HW5 
*/

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Function based on code provided in matmul.c
// Gets the time and is used for benchmarking
double gettime(void) {
  struct timeval tval;

  gettimeofday(&tval, NULL);

  return( (double)tval.tv_sec + (double)tval.tv_usec/1000000.0 );
}

// Kernel to compute the next generation
__global__ void next_generation_shared(int *grid, int *new_grid, int rows, int cols) {
    // Dynamically allocated shared memory
    extern __shared__ int shared_mem[];

    // Block and thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Global grid coordinates
    int global_x = bx * blockDim.x + tx;
    int global_y = by * blockDim.y + ty;

    // Shared memory dimensions
    int sh_width = blockDim.x + 2;
    int sh_height = blockDim.y + 2;

    // Shared memory 2D indexing macro
    #define SH_IDX(y, x) shared_mem[(y) * sh_width + (x)]

    // Load main cell and ghost cells
    if (global_x < cols && global_y < rows) {
        // Central cell
        SH_IDX(ty + 1, tx + 1) = grid[global_y * cols + global_x];

        // Ghost cell loading
        // Top row
        if (ty == 0) {
            if (global_y > 0)
                SH_IDX(0, tx + 1) = grid[(global_y - 1) * cols + global_x];
            
            // Top-left corner
            if (tx == 0 && global_x > 0 && global_y > 0)
                SH_IDX(0, 0) = grid[(global_y - 1) * cols + (global_x - 1)];
            
            // Top-right corner
            if (tx == blockDim.x - 1 && global_x < cols - 1 && global_y > 0)
                SH_IDX(0, tx + 2) = grid[(global_y - 1) * cols + (global_x + 1)];
        }

        // Bottom row
        if (ty == blockDim.y - 1) {
            if (global_y < rows - 1)
                SH_IDX(ty + 2, tx + 1) = grid[(global_y + 1) * cols + global_x];
            
            // Bottom-left corner
            if (tx == 0 && global_x > 0 && global_y < rows - 1)
                SH_IDX(ty + 2, 0) = grid[(global_y + 1) * cols + (global_x - 1)];
            
            // Bottom-right corner
            if (tx == blockDim.x - 1 && global_x < cols - 1 && global_y < rows - 1)
                SH_IDX(ty + 2, tx + 2) = grid[(global_y + 1) * cols + (global_x + 1)];
        }

        // Left column
        if (tx == 0 && global_x > 0)
            SH_IDX(ty + 1, 0) = grid[global_y * cols + (global_x - 1)];

        // Right column
        if (tx == blockDim.x - 1 && global_x < cols - 1)
            SH_IDX(ty + 1, tx + 2) = grid[global_y * cols + (global_x + 1)];
    }

    // Synchronize to ensure all shared memory is loaded
    __syncthreads();

    // Compute next state
    if (global_x >= 1 && global_x < cols - 1 && global_y >= 1 && global_y < rows - 1) {
        // Count live neighbors
        int neighbors = 
            SH_IDX(ty, tx) + 
            SH_IDX(ty, tx + 1) + 
            SH_IDX(ty, tx + 2) +
            SH_IDX(ty + 1, tx) + 
            SH_IDX(ty + 1, tx + 2) +
            SH_IDX(ty + 2, tx) + 
            SH_IDX(ty + 2, tx + 1) + 
            SH_IDX(ty + 2, tx + 2);

        // Apply Game of Life rules
        if (neighbors <= 1 || neighbors >= 4)
            new_grid[global_y * cols + global_x] = 0;  // Dies
        else if (neighbors == 3)
            new_grid[global_y * cols + global_x] = 1;  // Born
        else
            new_grid[global_y * cols + global_x] = SH_IDX(ty + 1, tx + 1);  // Stays the same
    }
}

// Function to initialize the grid with random values
void initialize_grid(int *grid, int rows, int cols) {
    srand(42);  // Fixed seed for reproducibility
    for (int i = 1; i < rows - 1; i++) {
        for (int j = 1; j < cols - 1; j++) {
            grid[i * cols + j] = rand() % 2;
        }
    }
}

// Output function
void outputtofile(char *output_file, int *grid, int rows, int cols) {
    FILE *file = fopen(output_file, "w");
    for (int i = 1; i < rows - 1; i++) {
        for (int j = 1; j < cols - 1; j++) {
            fprintf(file, "%i ", grid[i * cols + j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

// Function to get the current time in seconds
double get_time() {
    struct timeval tval;
    gettimeofday(&tval, NULL);
    return (double)tval.tv_sec + (double)tval.tv_usec / 1000000.0;
}

// Main function
int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Usage: %s <dimensions (int)> <max_generations (int)> <block size (NxN)> <stagnationcheck (boolean 1 or 0)> <output directory (string)>\n", argv[0]);
        exit(-1);
    }

    // Parse command line arguments
    int dimensions = atoi(argv[1]);
    int max_generations = atoi(argv[2]);
    int block_size = atoi(argv[3]);
    // Boolean for turning on and off stagnation check
    int stagnationcheck = atoi(argv[5]);

    int rows = dimensions + 2;  // Adding ghost rows
    int cols = dimensions + 2;

    size_t grid_size = rows * cols * sizeof(int);

    // Allocate memory for grids on host
    int *host_grid = (int *)malloc(grid_size);
    int *host_new_grid = (int *)malloc(grid_size);

    // Initialize the grid
    initialize_grid(host_grid, rows, cols);

    // Doubles to hold start and end time for benchmarking
    double starttime, endtime;

    // Allocate memory for grids on device
    int *dev_grid, *dev_new_grid;
    hipMalloc((void **)&dev_grid, grid_size);
    hipMalloc((void **)&dev_new_grid, grid_size);

    // Copy initial grid to device
    hipMemcpy(dev_grid, host_grid, grid_size, hipMemcpyHostToDevice);

    // Set up block and grid dimensions
    dim3 block_dim(block_size, block_size);
    dim3 grid_dim((cols + block_size - 1) / block_size, (rows + block_size - 1) / block_size);

    // Getting start time for benchmarking
    starttime = gettime();

    // Main simulation loop
    for (int gen = 0; gen < max_generations; gen++) {
        size_t shared_mem_size = (block_size + 2) * (block_size + 2) * sizeof(int);
        next_generation_shared<<<grid_dim, block_dim, shared_mem_size>>>(dev_grid, dev_new_grid, rows, cols);
   
        // Swap grids
        int *temp = dev_grid;
        dev_grid = dev_new_grid;
        dev_new_grid = temp;

        // Optional: Check for stagnation (if enabled)
        if (stagnationcheck) {
            // Add stagnation check logic here if required.
        }
    }

    // Copy final grid back to host
    hipMemcpy(host_grid, dev_grid, grid_size, hipMemcpyDeviceToHost);

    // Getting endtime and getting benchmarks
    endtime = gettime();
    printf("Time taken = %lf seconds\n", endtime-starttime);

    // Output file and directory (format output_N_N_gen_threads.txt)
    char output_file[200];
    sprintf(output_file, "%s/output%s_%s_%s.txt", argv[4], argv[1], argv[2], argv[3]);
    outputtofile(output_file, host_grid, rows, cols);

    // Free memory on device
    hipFree(dev_grid);
    hipFree(dev_new_grid);

    // Free memory on host
    free(host_grid);
    free(host_new_grid);

    return 0;
}
